﻿#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>
#include <cstdint>


#define CUDA_CHECK(x) do { \
    hipError_t err_ = (x); \
    if (err_ != hipSuccess) { \
        fprintf(stderr, "CUDA error %s at %s:%d\n", hipGetErrorString(err_), __FILE__, __LINE__); \
        std::abort(); \
    } \
} while(0)


// Extern declarations for Step 2 kernels
extern __global__ void fast_score_kernel(const uint8_t*, const uint8_t*, int,int,int,int,float,float*);
extern __global__ void nms3x3_kernel(const float*, int,int, float, uint2*, int, int*);


struct FrameBuffers {
    // Device
    uint8_t *d_img=nullptr, *d_msk=nullptr; float *d_scores=nullptr;
    uint2 *d_kp=nullptr;
    int *d_cnt=nullptr;
    // Host pinned
    uint8_t *h_img=nullptr, *h_msk=nullptr; int *h_cnt=nullptr;
    int W=0,H=0,max_kp=0; hipStream_t stream=0; hipEvent_t done;
};


extern "C" void step4_allocate(FrameBuffers& fb, int W, int H, int max_kp){
    fb.W=W;
    fb.H=H;
    fb.max_kp=max_kp;

    CUDA_CHECK(hipHostAlloc(&fb.h_img, W*H, hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc(&fb.h_msk, W*H, hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc(&fb.h_cnt, sizeof(int), hipHostMallocDefault));
    CUDA_CHECK(hipMalloc(&fb.d_img, W*H));
    CUDA_CHECK(hipMalloc(&fb.d_msk, W*H));
    CUDA_CHECK(hipMalloc(&fb.d_scores, W*H*sizeof(float)));
    CUDA_CHECK(hipMalloc(&fb.d_kp, max_kp*sizeof(uint2)));
    CUDA_CHECK(hipMalloc(&fb.d_cnt, sizeof(int)));
    CUDA_CHECK(hipStreamCreateWithFlags(&fb.stream, hipStreamNonBlocking));
    CUDA_CHECK(hipEventCreateWithFlags(&fb.done, hipEventDisableTiming));
}


extern "C" void step4_process_async(FrameBuffers& fb){
    // H->D async copies
    CUDA_CHECK(hipMemcpyAsync(fb.d_img, fb.h_img, fb.W*fb.H, hipMemcpyHostToDevice, fb.stream));
    CUDA_CHECK(hipMemcpyAsync(fb.d_msk, fb.h_msk, fb.W*fb.H, hipMemcpyHostToDevice, fb.stream));
    CUDA_CHECK(hipMemsetAsync(fb.d_cnt, 0, sizeof(int), fb.stream));

    dim3 BS(16,16), GS((fb.W+15)/16,(fb.H+15)/16);
    fast_score_kernel<<<GS,BS,0,fb.stream>>>(fb.d_img, fb.d_msk, fb.W, fb.H, fb.W, 20, 0.3f, fb.d_scores);
    size_t shmem=(16+2)*(16+2)*sizeof(float);
    nms3x3_kernel<<<GS,BS,shmem,fb.stream>>>(fb.d_scores, fb.W, fb.H, 5.0f, fb.d_kp, fb.max_kp, fb.d_cnt);

    // D->H async copy of count only (results buffer can remain device-resident for downstream)
    CUDA_CHECK(hipMemcpyAsync(fb.h_cnt, fb.d_cnt, sizeof(int), hipMemcpyDeviceToHost, fb.stream));
    CUDA_CHECK(hipEventRecord(fb.done, fb.stream));
}


extern "C" bool step4_poll_complete(FrameBuffers& fb){ return hipEventQuery(fb.done) == hipSuccess; }
extern "C" int step4_get_count(const FrameBuffers& fb){ return *fb.h_cnt; }
extern "C" void step4_free(FrameBuffers& fb){
    if(fb.d_img) hipFree(fb.d_img); if(fb.d_msk) hipFree(fb.d_msk); if(fb.d_scores) hipFree(fb.d_scores);
    if(fb.d_kp) hipFree(fb.d_kp); if(fb.d_cnt) hipFree(fb.d_cnt);
    if(fb.h_img) hipHostFree(fb.h_img); if(fb.h_msk) hipHostFree(fb.h_msk); if(fb.h_cnt) hipHostFree(fb.h_cnt);
    if(fb.stream) hipStreamDestroy(fb.stream); if(fb.done) hipEventDestroy(fb.done);
    fb = FrameBuffers{};
}